
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <chrono>

int ***Image;

void AllocateMemory(int n);
void FreeMemory(int n);
void InitImage(int n);
void WritePGM(const char *fname, int n, int index);
void GameOfLifeGPU(int n, int stepLimit);

int main(int argc, char *argv[]) {

    int imgSize = atoi(argv[1]) + 2; // +2 dla uproszeczenia warunkow
    int stepLimit = atoi(argv[2]);

    AllocateMemory(imgSize);
    InitImage(imgSize);
    
    auto start_time = std::chrono::high_resolution_clock::now();

    GameOfLifeGPU(imgSize, stepLimit);

    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

    printf("Execution time: %d msecs\n", duration.count());

    WritePGM("final.pgm", imgSize, (stepLimit-1)%2);

    FreeMemory(imgSize);

    //MPI_Finalize();

    return 0;
}

void AllocateMemory(int n) {
    Image = new int **[2];
    for (int i = 0; i < 2; ++i) {
        Image[i] = new int *[n];
        for (int j = 0; j < n; ++j) {
            Image[i][j] = new int[n];
        }
    }
}

void InitImage(int n){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            Image[0][i][j] = 0;
        }
    }

    Image[0][1][7] = 1;
    Image[0][2][1] = 1;
    Image[0][2][2] = 1;
    Image[0][3][2] = 1;
    Image[0][3][6] = 1;
    Image[0][3][7] = 1;
    Image[0][3][8] = 1;
}

void FreeMemory(int n) {
    for (int i = 0; i < 2; ++i) {
        for (int j = 0; j < n; ++j) {
            delete[] Image[i][j];
        }
        delete[] Image[i];
    }
    delete[] Image;
}


void WritePGM(const char *fname, int n, int index) {
    FILE *file = fopen(fname, "wt");
    if (!file) {
        perror("Error opening file");
        return;
    }

    fprintf(file, "P3\n%d %d\n255\n", n, n);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (Image[index][i][j] == 1) {
                fprintf(file, "255 0 0 ");
            } else {
                fprintf(file, "255 255 255 ");
            }
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

__global__ void GOLKernel(int* d_In, int* d_Out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i * n + j;

    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        int neighbors = 0;

        int i_minus_1 = (i - 1) * n;
        int i_plus_1 = (i + 1) * n;

        if (d_In[i_minus_1 + j - 1]) neighbors++;
        if (d_In[i_minus_1 + j]) neighbors++;
        if (d_In[i_minus_1 + j + 1]) neighbors++;
        if (d_In[idx - 1]) neighbors++;
        if (d_In[idx + 1]) neighbors++;
        if (d_In[i_plus_1 + j - 1]) neighbors++;
        if (d_In[i_plus_1 + j]) neighbors++;
        if (d_In[i_plus_1 + j + 1]) neighbors++;

        if (d_In[idx]) {
            if (neighbors > 3) {
                d_Out[idx] = 0;
            } else if (neighbors < 2) {
                d_Out[idx] = 0;
            } else {
                d_Out[idx] = 1;
            }
        } else {
            if (neighbors == 3) {
                d_Out[idx] = 1;
            } else {
                d_Out[idx] = 0;
            }
        }
    }
}

void GameOfLifeGPU(int n, int stepLimit) {

    int *d_Image, *d_outputImage;
    hipMalloc((void **)&d_Image, n * n * sizeof(int *));
    hipMalloc((void **)&d_outputImage, n * n * sizeof(int *));

    for (int step = 1; step < stepLimit; ++step) {

        int actualIndex = step & 1;
        int prevIndex = 1 - actualIndex;

        int* h_Image = new int[n * n];
        int* h_outputImage = new int[n * n];

        for (int i = 0; i < n; i++) {
            hipMemcpy(h_Image + i * n, Image[prevIndex][i], n * sizeof(int), hipMemcpyHostToHost);
        }

        hipMemcpy(d_Image, h_Image, n * n * sizeof(int), hipMemcpyHostToDevice);

        delete[] h_Image;

        dim3 blockSize(16, 16);
        dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

        GOLKernel <<<gridSize, blockSize>>> (d_Image, d_outputImage, n);

        hipMemcpy(h_outputImage, d_outputImage, n * n * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < n; ++i) {
            hipMemcpy(Image[actualIndex][i], h_outputImage + i * n, n * sizeof(int), hipMemcpyHostToHost);
        }
        
        delete[] h_outputImage;
    }

    hipFree(d_Image);
    hipFree(d_outputImage);

    hipDeviceSynchronize();
}
